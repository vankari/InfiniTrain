#include "hip/hip_runtime.h"
#include "infini_train/include/common/cuda/common_cuda.cuh"

namespace infini_train::kernels::cuda {

template <typename T>
__global__ void TrilForwardKernel(const T *input, T *output, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal >= 0) {
        output[idx] = input[idx];
    } else {
        output[idx] = T(0);
    }
}

std::shared_ptr<Tensor> TrilForward(const std::shared_ptr<Tensor> &input, int64_t diagonal) {
    CHECK_EQ(input->Dims().size(), 2);
    int64_t rows = input->Dims()[0];
    int64_t cols = input->Dims()[1];

    auto output = std::make_shared<Tensor>(input->Dims(), input->Dtype(), input->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(input->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        input->Dtype(),
        [=]<typename T>() {
            TrilForwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<T *>(input->DataPtr()), static_cast<T *>(output->DataPtr()), rows, cols, diagonal);
        },
        "CUDA TrilForward");

    return output;
}

template <typename T>
__global__ void TrilBackwardKernel(const T *grad_output, T *grad_input, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal >= 0) {
        grad_input[idx] = grad_output[idx];
    } else {
        grad_input[idx] = T(0);
    }
}

std::shared_ptr<Tensor> TrilBackward(const std::shared_ptr<Tensor> &grad_output, int64_t diagonal) {
    int rows = grad_output->Dims()[0];
    int cols = grad_output->Dims()[1];

    auto dtype = grad_output->Dtype();
    auto grad_input = std::make_shared<Tensor>(grad_output->Dims(), dtype, grad_output->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(grad_output->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        dtype,
        [=]<typename T>() {
            grad_input->Fill<T>(0);
            TrilBackwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(grad_output->DataPtr()), static_cast<T *>(grad_input->DataPtr()), rows, cols,
                diagonal);
        },
        "CUDA TrilBackward");

    return grad_input;
}

template <typename T>
__global__ void TriuForwardKernel(const T *input, T *output, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal <= 0) {
        output[idx] = input[idx];
    } else {
        output[idx] = T(0);
    }
}

std::shared_ptr<Tensor> TriuForward(const std::shared_ptr<Tensor> &input, int64_t diagonal) {
    CHECK_EQ(input->Dims().size(), 2);
    int64_t rows = input->Dims()[0];
    int64_t cols = input->Dims()[1];

    auto output = std::make_shared<Tensor>(input->Dims(), input->Dtype(), input->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(input->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        input->Dtype(),
        [=]<typename T>() {
            TriuForwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(input->DataPtr()), static_cast<T *>(output->DataPtr()), rows, cols, diagonal);
        },
        "CUDA TriuForward");

    return output;
}

template <typename T>
__global__ void TriuBackwardKernel(const T *grad_output, T *grad_input, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal <= 0) {
        grad_input[idx] = grad_output[idx];
    } else {
        grad_input[idx] = T(0);
    }
}

std::shared_ptr<Tensor> TriuBackward(const std::shared_ptr<Tensor> &grad_output, int64_t diagonal) {
    int rows = grad_output->Dims()[0];
    int cols = grad_output->Dims()[1];

    auto dtype = grad_output->Dtype();
    auto grad_input = std::make_shared<Tensor>(grad_output->Dims(), dtype, grad_output->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;
    const auto *cuda_device = dynamic_cast<const CudaDevice *>(grad_output->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        dtype,
        [=]<typename T>() {
            grad_input->Fill<T>(0);
            TriuBackwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(grad_output->DataPtr()), static_cast<T *>(grad_input->DataPtr()), rows, cols,
                diagonal);
        },
        "CUDA TriuBackward");

    return grad_input;
}

template <typename T>
__global__ void TransposeForwardKernel(const T *input, T *output, const int64_t *in_dims, const int64_t *in_strides,
                                       const int64_t *out_strides, int64_t ndim, int64_t dim0, int64_t dim1,
                                       int64_t num_elements) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_elements) {
        return;
    }

    int64_t remaining = idx;
    // TODO(zbl): assume ndim <= 8 here
    int64_t coords[8];

    // 1. decode coord from output index
    for (int i = 0; i < ndim; ++i) {
        coords[i] = remaining / out_strides[i];
        remaining %= out_strides[i];
    }

    // 2. swap the coordinates
    int64_t tmp = coords[dim0];
    coords[dim0] = coords[dim1];
    coords[dim1] = tmp;

    // 3. compute input flat index
    int64_t in_flat_idx = 0;
    for (int i = 0; i < ndim; ++i) { in_flat_idx += coords[i] * in_strides[i]; }

    output[idx] = input[in_flat_idx];
}

std::shared_ptr<Tensor> TransposeForward(const std::shared_ptr<Tensor> &input, int64_t dim0, int64_t dim1) {
    // TODO(zbl): assume ndim <= 8 here
    CHECK_LE(input->Dims().size(), 8);
    dim0 = dim0 < 0 ? dim0 + input->Dims().size() : dim0;
    dim1 = dim1 < 0 ? dim1 + input->Dims().size() : dim1;
    CHECK(dim0 >= 0 && dim0 < input->Dims().size() && dim1 >= 0 && dim1 < input->Dims().size());

    auto in_dims = input->Dims();
    std::vector<int64_t> out_dims = in_dims;
    std::swap(out_dims[dim0], out_dims[dim1]);

    auto dtype = input->Dtype();
    auto output = std::make_shared<Tensor>(out_dims, dtype, input->GetDevice());
    int64_t ndim = in_dims.size();
    int64_t num_elements = output->NumElements();

    // compute strides of in_dims and out_dims
    std::vector<int64_t> in_strides(ndim, 1);
    std::vector<int64_t> out_strides(ndim, 1);
    for (int i = ndim - 2; i >= 0; --i) {
        in_strides[i] = in_strides[i + 1] * in_dims[i + 1];
        out_strides[i] = out_strides[i + 1] * out_dims[i + 1];
    }

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(input->GetDevice());
    const auto &stream = cuda_device->Stream();

    // Allocate device memory for dims and strides
    // TODO(zbl): avoid using hipMalloc?
    int64_t *device_buffer;
    hipMallocAsync(&device_buffer, 3 * ndim * sizeof(int64_t), stream);

    int64_t *in_dims_dev = device_buffer;
    int64_t *in_strides_dev = device_buffer + ndim;
    int64_t *out_strides_dev = device_buffer + 2 * ndim;

    std::vector<int64_t> host_buffer;
    host_buffer.insert(host_buffer.end(), in_dims.begin(), in_dims.end());
    host_buffer.insert(host_buffer.end(), in_strides.begin(), in_strides.end());
    host_buffer.insert(host_buffer.end(), out_strides.begin(), out_strides.end());

    hipMemcpyAsync(device_buffer, host_buffer.data(), 3 * ndim * sizeof(int64_t), hipMemcpyHostToDevice, stream);

    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    DispatchFunc<INFINI_ALL_TYPES>(
        dtype,
        [=]<typename T>() {
            output->Fill<T>(0);
            TransposeForwardKernel<<<num_blocks, threads_per_block, 0, stream>>>(
                static_cast<const T *>(input->DataPtr()), static_cast<T *>(output->DataPtr()), in_dims_dev,
                in_strides_dev, out_strides_dev, ndim, dim0, dim1, num_elements);
        },
        "CUDA TransposeForward");

    hipFreeAsync(device_buffer, stream);

    return output;
}

std::shared_ptr<Tensor> TransposeBackward(const std::shared_ptr<Tensor> &grad_output, int64_t dim0, int64_t dim1) {
    return TransposeForward(grad_output, dim1, dim0);
}

template <typename T>
__global__ void MaskForwardKernel(const T *input, const T *mask, T *output, T value, int batch_size, int mask_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < batch_size * mask_size) {
        output[i] = (mask[i % mask_size] == T(1)) ? value : input[i];
    }
}

std::shared_ptr<Tensor> MaskForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &mask,
                                    float value) {
    auto input_shape = input->Dims();
    auto mask_shape = mask->Dims();
    auto dtype = input->Dtype();
    CHECK_EQ(static_cast<int>(dtype), static_cast<int>(mask->Dtype()));

    int64_t input_dims = input_shape.size();
    int64_t mask_dims = mask_shape.size();
    for (int i = 0; i < mask_dims; ++i) {
        int input_dim = input_shape[input_dims - mask_dims + i];
        int mask_dim = mask_shape[i];
        CHECK(input_dim == mask_dim || mask_dim == 1);
    }

    int64_t mask_size = mask->NumElements();
    int64_t batch_size = input->NumElements() / mask_size;

    auto output = std::make_shared<Tensor>(input->Dims(), dtype, input->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (input->NumElements() + threads_per_block - 1) / threads_per_block;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(output->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        dtype,
        [=]<typename T>() {
            MaskForwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(input->DataPtr()), static_cast<const T *>(mask->DataPtr()),
                static_cast<T *>(output->DataPtr()), common::cuda::Cast<T>(value), batch_size, mask_size);
        },
        "CUDA MaskForward");

    return output;
}

template <typename T>
__global__ void MaskBackwardKernel(const T *grad_output, const T *mask, T *grad_input, int batch_size, int mask_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < batch_size * mask_size) {
        grad_input[i] = (mask[i % mask_size] == T(1)) ? T(0) : grad_output[i];
    }
}

std::shared_ptr<Tensor> MaskBackward(const std::shared_ptr<Tensor> &grad_output, const std::shared_ptr<Tensor> &mask) {
    auto output_shape = grad_output->Dims();
    auto mask_shape = mask->Dims();
    auto dtype = grad_output->Dtype();
    auto grad_output_ = std::make_shared(grad_output->To(DataType::kFLOAT32));
    CHECK_EQ(static_cast<int>(grad_output_->Dtype()), static_cast<int>(mask->Dtype()));

    int64_t output_dims = output_shape.size();
    int64_t mask_dims = mask_shape.size();
    for (int i = 0; i < mask_dims; ++i) {
        int out_dim = output_shape[output_dims - mask_dims + i];
        int mask_dim = mask_shape[i];
        CHECK(out_dim == mask_dim || mask_dim == 1);
    }

    int64_t mask_size = mask->NumElements();
    int64_t batch_size = grad_output->NumElements() / mask_size;

    auto grad_input = std::make_shared<Tensor>(grad_output_->Dims(), grad_output_->Dtype(), grad_output_->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (grad_output->NumElements() + threads_per_block - 1) / threads_per_block;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(grad_output_->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        dtype,
        [=]<typename T>() {
            grad_input->Fill<T>(0);
            MaskBackwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(grad_output_->DataPtr()), static_cast<const T *>(mask->DataPtr()),
                static_cast<T *>(grad_input->DataPtr()), batch_size, mask_size);
        },
        "CUDA MaskBackward");

    return grad_input;
}

template <typename T>
__global__ void RepeatInterleaveForwardKernel(const T *input, T *output, int64_t outer, int64_t dim_size, int64_t inner,
                                              int64_t repeat) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t total = outer * dim_size * repeat * inner;
    if (idx >= total) {
        return;
    }

    int64_t i = idx / inner;
    int64_t j = idx % inner;

    int64_t o = i / (dim_size * repeat);
    int64_t di = (i / repeat) % dim_size;

    output[idx] = input[(o * dim_size + di) * inner + j];
}

std::shared_ptr<Tensor> RepeatInterleaveForward(const std::shared_ptr<Tensor> &input, int64_t repeat, int64_t dim) {
    CHECK_GT(repeat, 0);
    CHECK_GE(dim, 0);
    CHECK_LT(dim, input->Dims().size());

    const auto &input_dims = input->Dims();
    const int64_t outer = std::accumulate(input_dims.begin(), input_dims.begin() + dim, 1, std::multiplies<int64_t>());
    const int64_t inner
        = std::accumulate(input_dims.begin() + dim + 1, input_dims.end(), 1, std::multiplies<int64_t>());
    const int64_t dim_size = input_dims[dim];

    std::vector<int64_t> output_dims = input_dims;
    output_dims[dim] = dim_size * repeat;
    auto output = std::make_shared<Tensor>(output_dims, input->Dtype(), input->GetDevice());

    int64_t total_elements = outer * dim_size * repeat * inner;
    int threads_per_block = 256;
    int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;
    const auto *cuda_device = dynamic_cast<const CudaDevice *>(input->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        input->Dtype(),
        [=]<typename T>() {
            RepeatInterleaveForwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(input->DataPtr()), static_cast<T *>(output->DataPtr()), outer, dim_size, inner,
                repeat);
        },
        "CUDA RepeatInterleaveForward");

    return output;
}

template <typename T>
__global__ void RepeatInterleaveBackwardKernel(const T *grad_output, T *grad_input, int64_t outer, int64_t dim_size,
                                               int64_t inner, int64_t repeat) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t total = outer * dim_size * inner;
    if (idx >= total) {
        return;
    }

    int64_t i = idx / inner;
    int64_t j = idx % inner;

    int64_t o = i / dim_size;
    int64_t di = i % dim_size;

    T sum = T(0);
    for (int64_t r = 0; r < repeat; ++r) {
        int64_t out_idx = ((o * dim_size * repeat + di * repeat + r) * inner) + j;
        sum += grad_output[out_idx];
    }
    grad_input[idx] = sum;
}

std::shared_ptr<Tensor> RepeatInterleaveBackward(const std::shared_ptr<Tensor> &grad_output,
                                                 const std::vector<int64_t> &input_dims, int64_t dim) {
    CHECK_GE(dim, 0);
    CHECK_LT(dim, input_dims.size());

    const int64_t outer = std::accumulate(input_dims.begin(), input_dims.begin() + dim, 1, std::multiplies<int64_t>());
    const int64_t inner
        = std::accumulate(input_dims.begin() + dim + 1, input_dims.end(), 1, std::multiplies<int64_t>());
    const int64_t dim_size = input_dims[dim];

    int64_t repeat = grad_output->Dims()[dim] / dim_size;
    CHECK_EQ(grad_output->Dims()[dim], dim_size * repeat);

    auto grad_input = std::make_shared<Tensor>(input_dims, grad_output->Dtype(), grad_output->GetDevice());

    int64_t total_elements = outer * dim_size * inner;
    int threads_per_block = 256;
    int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;
    const auto *cuda_device = dynamic_cast<const CudaDevice *>(grad_output->GetDevice());

    DispatchFunc<INFINI_ALL_TYPES>(
        grad_output->Dtype(),
        [=]<typename T>() {
            grad_input->Fill<T>(0);
            RepeatInterleaveBackwardKernel<<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(grad_output->DataPtr()), static_cast<T *>(grad_input->DataPtr()), outer,
                dim_size, inner, repeat);
        },
        "CUDA RepeatInterleaveBackward");

    return grad_input;
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_TRANSFORM_KERNEL(kernel_name)                                                                    \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_TRANSFORM_KERNEL(TrilForward)
REGISTER_CUDA_TRANSFORM_KERNEL(TrilBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(TriuForward)
REGISTER_CUDA_TRANSFORM_KERNEL(TriuBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(TransposeForward)
REGISTER_CUDA_TRANSFORM_KERNEL(TransposeBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(MaskForward)
REGISTER_CUDA_TRANSFORM_KERNEL(MaskBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(RepeatInterleaveForward)
REGISTER_CUDA_TRANSFORM_KERNEL(RepeatInterleaveBackward)

#undef REGISTER_CUDA_TRANSFORM_KERNEL
