#include "hip/hip_runtime.h"
#include <memory>
#include <tuple>

#include <cub/block/block_reduce.cuh>

#include "infini_train/include/common/cuda/common_cuda.h"
#include "infini_train/include/common/cuda/kernel_helper.cuh"
#include "infini_train/include/device.h"
#include "infini_train/include/dispatcher.h"
#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

template <int BLOCK_SIZE, typename T>
__global__ void LayerNormForwardKernel(const T *input, const T *weight, const T *bias, float *mean_out, float *rstd_out,
                                       T *output, float eps, int embed_dim) {
    using BlockReduce = hipcub::BlockReduce<float, BLOCK_SIZE>;
    __shared__ typename BlockReduce::TempStorage temp_storage_mean;
    __shared__ typename BlockReduce::TempStorage temp_storage_rstd;
    __shared__ float shared_mean;
    __shared__ float shared_rstd;

    const int token_idx = blockIdx.x;
    const T *x = input + token_idx * embed_dim;
    T *y = output + token_idx * embed_dim;

    float sum = 0.0f;
    float sqsum = 0.0f;

    for (int i = threadIdx.x; i < embed_dim; i += BLOCK_SIZE) {
        float val = common::cuda::Cast<float>(x[i]);
        sum += val;
        sqsum += val * val;
    }

    float total_sum = BlockReduce(temp_storage_mean).Sum(sum);
    float total_sqsum = BlockReduce(temp_storage_rstd).Sum(sqsum);

    if (threadIdx.x == 0) {
        float mean = total_sum / embed_dim;
        float var = total_sqsum / embed_dim - mean * mean;
        float rstd = rsqrtf(var + eps);
        shared_mean = mean;
        shared_rstd = rstd;
        if (mean_out) {
            mean_out[token_idx] = mean;
        }
        if (rstd_out) {
            rstd_out[token_idx] = rstd;
        }
    }
    __syncthreads();

    for (int i = threadIdx.x; i < embed_dim; i += BLOCK_SIZE) {
        float norm = (common::cuda::Cast<float>(x[i]) - shared_mean) * shared_rstd;
        y[i] = common::cuda::Cast<T>(norm * common::cuda::Cast<float>(weight[i]) + common::cuda::Cast<float>(bias[i]));
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LayerNormForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                 const std::shared_ptr<Tensor> &bias, const float eps) {
    CHECK_EQ(input->Dims().size(), 3);
    CHECK_LE(input->Dims()[2], weight->Dims()[0]);
    CHECK_LE(input->Dims()[2], bias->Dims()[0]);

    const int batch_size = input->Dims()[0];
    const int max_seqlen = input->Dims()[1];
    const int embed_dim = input->Dims()[2];

    auto dtype = input->Dtype();

    auto output = std::make_shared<Tensor>(input->Dims(), dtype, input->GetDevice());
    auto mean = std::make_shared<Tensor>(std::vector<int64_t>{batch_size, max_seqlen}, DataType::kFLOAT32,
                                         input->GetDevice());
    auto rstd = std::make_shared<Tensor>(std::vector<int64_t>{batch_size, max_seqlen}, DataType::kFLOAT32,
                                         input->GetDevice());

    constexpr int BLOCK_SIZE = 256;
    int threads_per_block = BLOCK_SIZE;
    int num_blocks = batch_size * max_seqlen;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(input->GetDevice());
    DispatchFunc<INFINI_ALL_FLOATING_TYPES>(
        dtype,
        [=]<typename T>() {
            mean->Fill<float>(0);
            rstd->Fill<float>(0);
            LayerNormForwardKernel<BLOCK_SIZE><<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(input->DataPtr()), static_cast<const T *>(weight->DataPtr()),
                static_cast<const T *>(bias->DataPtr()), static_cast<float *>(mean->DataPtr()),
                static_cast<float *>(rstd->DataPtr()), static_cast<T *>(output->DataPtr()), eps, embed_dim);
        },
        "CUDA LayerNormForward");

    return {output, mean, rstd};
}

template <int BLOCK_SIZE, typename T>
__global__ void LayerNormBackwardKernel(const T *__restrict__ input, const T *__restrict__ grad_output,
                                        const float *__restrict__ mean, const float *__restrict__ rstd,
                                        const T *__restrict__ weight, T *__restrict__ grad_input,
                                        T *__restrict__ grad_weight, T *__restrict__ grad_bias, int embed_dim,
                                        size_t weight_num_elements, size_t bias_num_elements) {
    using BlockReduce = hipcub::BlockReduce<float, BLOCK_SIZE>;
    __shared__ typename BlockReduce::TempStorage temp_storage_mean;
    __shared__ typename BlockReduce::TempStorage temp_storage_norm;
    __shared__ float shared_mean;
    __shared__ float shared_norm;

    int tid = threadIdx.x;
    int token_idx = blockIdx.x;

    const T *input_ptr = input + token_idx * embed_dim;
    const T *grad_output_ptr = grad_output + token_idx * embed_dim;
    T *grad_input_ptr = grad_input + token_idx * embed_dim;

    float mean_val = mean[token_idx];
    float rstd_val = rstd[token_idx];

    float dnorm_mean = 0.f;
    float dnorm_norm_mean = 0.f;

    for (int i = tid; i < embed_dim; i += BLOCK_SIZE) {
        float dnorm = common::cuda::Cast<float>(common::cuda::Mul(weight[i], grad_output_ptr[i]));
        dnorm_mean += dnorm;
        dnorm_norm_mean += dnorm * (common::cuda::Cast<float>(input_ptr[i]) - mean_val);
    }

    dnorm_mean = BlockReduce(temp_storage_mean).Sum(dnorm_mean);
    dnorm_norm_mean = BlockReduce(temp_storage_norm).Sum(dnorm_norm_mean);

    if (tid == 0) {
        float mean_d = dnorm_mean / embed_dim;
        float norm_d = (dnorm_norm_mean / embed_dim) * rstd_val - mean_d * mean_val * rstd_val;
        shared_mean = mean_d;
        shared_norm = norm_d;
    }
    __syncthreads();

    for (int i = tid; i < embed_dim; i += BLOCK_SIZE) {
        float norm = (common::cuda::Cast<float>(input_ptr[i]) - mean_val) * rstd_val;
        float grad_output_val = common::cuda::Cast<float>(grad_output_ptr[i]);

        grad_input_ptr[i] = common::cuda::Cast<T>(
            (common::cuda::Cast<float>(weight[i]) * grad_output_val - shared_mean - norm * shared_norm) * rstd_val);

        common::cuda::fastAtomicAdd<T, size_t>(grad_weight, i, weight_num_elements,
                                               common::cuda::Cast<T>(grad_output_val * norm), true);
        common::cuda::fastAtomicAdd<T, size_t>(grad_bias, i, bias_num_elements, grad_output_ptr[i], true);
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LayerNormBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                  const std::shared_ptr<Tensor> &bias, const std::shared_ptr<Tensor> &mean,
                  const std::shared_ptr<Tensor> &rstd, const std::shared_ptr<Tensor> &grad_output) {
    const int batch_size = input->Dims()[0];
    const int max_seqlen = input->Dims()[1];
    const int embed_dim = input->Dims()[2];

    auto dtype = input->Dtype();
    auto grad_input = std::make_shared<Tensor>(input->Dims(), dtype, grad_output->GetDevice());
    auto grad_weight = std::make_shared<Tensor>(weight->Dims(), dtype, grad_output->GetDevice());
    auto grad_bias = std::make_shared<Tensor>(bias->Dims(), dtype, grad_output->GetDevice());

    constexpr int BLOCK_SIZE = 256;
    int threads_per_block = BLOCK_SIZE;
    int num_blocks = batch_size * max_seqlen;

    const auto *cuda_device = dynamic_cast<const CudaDevice *>(input->GetDevice());
    DispatchFunc<DataType::kFLOAT32, DataType::kBFLOAT16>(
        dtype,
        [=]<typename T>() {
            grad_input->Fill<T>(0);
            grad_weight->Fill<T>(0);
            grad_bias->Fill<T>(0);
            LayerNormBackwardKernel<BLOCK_SIZE><<<num_blocks, threads_per_block, 0, cuda_device->Stream()>>>(
                static_cast<const T *>(input->DataPtr()), static_cast<const T *>(grad_output->DataPtr()),
                static_cast<const float *>(mean->DataPtr()), static_cast<const float *>(rstd->DataPtr()),
                static_cast<const T *>(weight->DataPtr()), static_cast<T *>(grad_input->DataPtr()),
                static_cast<T *>(grad_weight->DataPtr()), static_cast<T *>(grad_bias->DataPtr()), embed_dim,
                grad_weight->NumElements(), grad_bias->NumElements());
        },
        "CUDA LayerNormBackward");

    return {grad_input, grad_weight, grad_bias};
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_LAYERNORM_KERNEL(kernel_name)                                                                    \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_LAYERNORM_KERNEL(LayerNormForward)
REGISTER_CUDA_LAYERNORM_KERNEL(LayerNormBackward)

#undef REGISTER_CUDA_LAYERNORM_KERNEL
